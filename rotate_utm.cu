#include "hip/hip_runtime.h"
#include "rotate_utm.h"

#include <algorithm>
#include <assert.h>
#include <hip/hip_runtime.h>
#define NDEBUG
#include <cuproj/projection_factories.cuh>
#include <cuproj/vec_2d.hpp>
#include <errno.h>
#include <limits.h> // PATH_MAX
#include <math.h>
#include <npp.h>
#include <stddef.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>

#include "defs.h"
#include "nppdefs.h"
#include "nppi_geometry_transforms.h"
#include "rotate.h" // for get_lat_lon_min_max
#include "utils.h"

using namespace std;

extern long long mem_limit; // defined in main.c

struct rotate_utm_state {
        hipStream_t stream;
        NppStreamContext nppStreamCtx;
};

struct rotate_utm_state *rotate_utm_init(hipStream_t stream)
{
        struct rotate_utm_state *s = (struct rotate_utm_state *)calloc(
            1, sizeof *s);
        assert(s != NULL);
        s->stream = stream;

        init_npp_context(&s->nppStreamCtx, stream);

        return s;
}

void rotate_utm_destroy(struct rotate_utm_state *s)
{
        if (s == NULL) {
                return;
        }
        free(s);
}

/**
 * normalize the coordinates to 0..1 and
 * @return asoect ratio
 */
static double normalize_coords(const struct coordinate src_coords[4],
                               struct coordinate coords[4])
{
        double lat_min = 0;
        double lat_max = 0;
        double lon_min = 0;
        double lon_max = 0;

        // check if not near poles
        get_lat_lon_min_max(src_coords, &lat_min, &lat_max, &lon_min, &lon_max);
        if (lat_min < -85. || lat_max > 85.) {
                const double near_pole_pt_lat = (lat_max > 85 ? lat_max
                                                              : lat_min);
                WARN_MSG("Not normalizing areas near North/South Pole! (at "
                         "least one point at most 5° /%f°/ degrees from the "
                         "Pole)\n",
                         near_pole_pt_lat);
                return -1;
        }

        for (unsigned i = 0; i < 4; ++i) {
                double lat_rad = src_coords[i].latitude / 180. * M_PI;
                coords[i].latitude = (M_PI -
                                      log(tan((M_PI / 4.) + (lat_rad / 2.)))) /
                                     (2. * M_PI);
                double lon_rad = src_coords[i].longitude / 180. * M_PI;
                coords[i].longitude = (M_PI + lon_rad) / (2. * M_PI);
        }

        get_lat_lon_min_max(coords, &lat_min, &lat_max, &lon_min, &lon_max);

        double lat_range = lat_max - lat_min;
        double lon_range = lon_max - lon_min;

        // fprintf(stderr,
        //         "lat_min: %f lat_max: %f lon_min: %f lon_max: %f lat_range: %f "
        //         "lon_range: %f\n",
        //         lat_min, lat_max, lon_min, lon_max, lat_range, lon_range);

        for (unsigned i = 0; i < 4; ++i) {
                coords[i].latitude = (coords[i].latitude - lat_min) / lat_range;
                coords[i].longitude = (coords[i].longitude - lon_min) /
                                      lon_range;
        }

        return lon_range / lat_range;
}

/// fullfill GPUJPEG mem requirements
static void adjust_size(int *width, int *height, int comp_count) {

        enum {
                GB1 = 1LL * 1000 * 1000 * 1000,
                GJ_PER_BYTE_REQ = 20,
        };
        ssize_t threshold = mem_limit;
        if (threshold == 0) {
                threshold = MIN((ssize_t)gpu_memory / 2,
                                (ssize_t)gpu_memory - 2 * GB1);
                assert(threshold >= (ssize_t)2 * GB1);
        }
        ssize_t gj_gram_needed = (ssize_t)*width * *height * comp_count *
                                 GJ_PER_BYTE_REQ;
        if (gj_gram_needed < threshold) {
                return;
        }
        WARN_MSG(
            "[rotate] Encoding of %dx%d image would require %.2f GB GRAM (>=%g "
            "GB), downsizing ",
            *width, *height, (double)gj_gram_needed / GB1,
            (double)threshold / GB1);
        while (gj_gram_needed > threshold) {
                *width /= 2;
                *height /= 2;
                gj_gram_needed /= 4;
        }
        WARN_MSG("rotated to %.2f GB (%dx%d).\n", (double)gj_gram_needed / GB1,
                 *width, *height);
}

static void release_owned_image(struct owned_image *img) {
        CHECK_CUDA(hipFree(img->img.data));
        free(img);
}

struct bounds {
        float bound[4];
};
using device_projection = cuproj::device_projection<cuproj::vec_2d<float>>;
static __global__ void kernel(device_projection const d_proj,device_projection const d_proj2,
                              const uint8_t *d_in, uint8_t *d_out, int in_width,
                              int in_height, int out_width, int out_height,
                              struct bounds src_bounds,
                              struct bounds dst_bounds)
{
        int x = blockIdx.x * blockDim.x + threadIdx.x; // column index
        int y = blockIdx.y * blockDim.y + threadIdx.y; // row index

        if (x >= out_width || y >= out_height) {
                return;
        }

        float lat_scale = dst_bounds.bound[YMAX] - dst_bounds.bound[YMIN];
        float this_lat = dst_bounds.bound[YMIN];
        this_lat += lat_scale * ((x + .5f) / out_height);

        float lon_scale = dst_bounds.bound[XMAX] - dst_bounds.bound[XMIN];
        float this_lon = dst_bounds.bound[XMIN];
        this_lon += lon_scale * ((y + .5f) / out_width);

        if (x > 0 || y > 0) return;
        cuproj::vec_2d<float> in{500000,0};
        printf("\t%f ddd  %f\n\n\n\n", this_lat, this_lon);
        printf("\t%f ddd %f\n\n\n\n", in.x, in.y);
        cuproj::vec_2d<float> out = d_proj.transform(in);
        printf("%f %f\n", out.x, out.y);
        in = d_proj2.transform(out);
        printf("%f %f\n", in.x, in.y);
        
        d_out[x + y * out_width] = d_in[x + y * in_width];
}

__global__ void example_kernel(device_projection const d_proj,
                               cuproj::vec_2d<float> const* in,
                               cuproj::vec_2d<float>* out,
                               size_t n)
{
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x) {
    out[i] = d_proj.transform(in[i]);
  }
}

void test() {
  using coordinate = cuproj::vec_2d<float>;

  // Make a projection to convert WGS84 (lat, lon) coordinates to
  // UTM zone 56S (x, y) coordinates
  auto proj = cuproj::make_projection<coordinate>("EPSG:4326", "EPSG:32756");

  // Sydney, NSW, Australia
  coordinate sydney{-33.858700, 151.214000};
  thrust::device_vector<coordinate> d_in{1, sydney};
  thrust::device_vector<coordinate> d_out(d_in.size());

  auto d_proj            = proj->get_device_projection(cuproj::direction::FORWARD);
  std::size_t block_size = 256;
  std::size_t grid_size  = (d_in.size() + block_size - 1) / block_size;
  example_kernel<<<grid_size, block_size>>>(
    d_proj, d_in.data().get(), d_out.data().get(), d_in.size());
  hipDeviceSynchronize();
        CHECK_CUDA(hipDeviceSynchronize());
        exit(1);
        
}

static struct owned_image *to_epsg_4326(struct rotate_utm_state *s,
                                        const struct dec_image *in)
{
        // test(); return nullptr;
        double src_ratio = (in->bounds[XMAX] - in->bounds[XMIN]) /
                           (in->bounds[YMAX] - in->bounds[YMIN]);
        double lat_top = max(in->coords[ULEFT].latitude, in->coords[URIGHT].latitude);
        double lat_bot = min(in->coords[BLEFT].latitude, in->coords[BRIGHT].latitude);
        double lon_left = min(in->coords[ULEFT].longitude, in->coords[BLEFT].longitude);
        double lon_right = max(in->coords[URIGHT].longitude, in->coords[BRIGHT].longitude);
        double dst_ratio = (lon_right - lon_left) / (lat_top - lat_bot);
        struct bounds dst_bounds;
        dst_bounds.bound[XMIN] = lon_left;
        dst_bounds.bound[YMAX] = lat_top;
        dst_bounds.bound[XMAX] = lon_right;
        dst_bounds.bound[YMIN] = lat_bot;
        struct bounds src_bounds;
        src_bounds.bound[XMIN] = in->bounds[XMIN];
        src_bounds.bound[YMAX] = in->bounds[YMAX];
        src_bounds.bound[XMAX] = in->bounds[XMAX];
        src_bounds.bound[YMIN] = in->bounds[YMIN];
        struct dec_image dst_desc = *in;
        if (dst_ratio >= src_ratio) {
                dst_desc.width = (int)(in->height * dst_ratio);
        } else {
                dst_desc.height = (int)(in->width / dst_ratio);
        }
        struct owned_image *ret = new_cuda_owned_image(&dst_desc);

        using coordinate = cuproj::vec_2d<float>;
        auto proj = cuproj::make_projection<coordinate>(in->authority, "EPSG:4326"
                                                         );
        auto d_proj = proj->get_device_projection(cuproj::direction::FORWARD);
        auto d_proj2 = proj->get_device_projection(cuproj::direction::INVERSE);

        dim3 block(16, 16);
        int width = dst_desc.width;
        int height = dst_desc.height;
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
        kernel<<<grid, block, 0, s->stream>>>(d_proj,d_proj2,
            in->data, ret->img.data, in->width, in->height, width, height, src_bounds, dst_bounds);

        return ret;
}

struct owned_image *rotate_utm(struct rotate_utm_state *s, const struct dec_image *in)
{
        struct owned_image *epsg4326 = to_epsg_4326(s, in);
        if (epsg4326 == nullptr) {
                return nullptr;
        }
        return epsg4326;

        double aSrcQuad[4][2] = {
            {0.0, 0.0},                              // Top-left
            {(double)in->width, 0},                  // Top-right
            {(double)in->width, (double)in->height}, // Bottom-right
            {0.0, (double)in->height}                // Bottom-left
        };

        struct coordinate coords[4];
        const double dst_aspect = normalize_coords(in->coords, coords);

        NppiRect oSrcROI = {0, 0, in->width, in->height};
        NppiSize oSrcSize = {in->width, in->height};

        struct owned_image *ret = (struct owned_image *) malloc(sizeof *ret);
        memcpy(&ret->img, in, sizeof *in);
        ret->free = release_owned_image;
        // keep one side as in original and upscale the other to meet dst
        // projection dimension
        const double src_aspect = (double)in->width / in->height;
        if (dst_aspect >= src_aspect) {
                ret->img.width = (int)(ret->img.height * dst_aspect);
        } else {
                ret->img.height = (int)(ret->img.width / dst_aspect);
        }
        adjust_size(&ret->img.width, &ret->img.height, in->comp_count);

        const size_t req_size = (size_t)ret->img.width * ret->img.height *
                                ret->img.comp_count;
        CHECK_CUDA(hipMalloc((void **)&ret->img.data, req_size));

        NppiRect oDstROI = {0, 0, ret->img.width, ret->img.height};
        double aDstQuad[4][2] = {
            {coords[0].longitude * ret->img.width,
             coords[0].latitude * ret->img.height}, // Top-left
            {coords[1].longitude * ret->img.width,
             coords[1].latitude * ret->img.height}, // Top-right
            {coords[2].longitude * ret->img.width,
             coords[2].latitude * ret->img.height}, // Bottom-right
            {coords[3].longitude * ret->img.width,
             coords[3].latitude * ret->img.height}, // Bottom-left
        };

        GPU_TIMER_START(rotate, LL_DEBUG, s->stream);
        CHECK_CUDA(hipMemsetAsync(ret->img.data, 0,
                                   (size_t)ret->img.width * ret->img.height *
                                       ret->img.comp_count,
                                   s->stream));
        const int interpolation = NPPI_INTER_LINEAR;
        if (in->comp_count == 1) {
                CHECK_NPP(nppiWarpPerspectiveQuad_8u_C1R_Ctx(
                    in->data, oSrcSize, in->width, oSrcROI, aSrcQuad,
                    ret->img.data, ret->img.width, oDstROI, aDstQuad,
                    interpolation, s->nppStreamCtx));
        } else {
                assert(in->comp_count == 3);
                CHECK_NPP(nppiWarpPerspectiveQuad_8u_C3R_Ctx(
                    in->data, oSrcSize, 3 * in->width, oSrcROI, aSrcQuad,
                    ret->img.data, 3 * ret->img.width, oDstROI, aDstQuad,
                    interpolation, s->nppStreamCtx));
        }
        GPU_TIMER_STOP(rotate);

        return ret;
}
