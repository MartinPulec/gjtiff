#include "hip/hip_runtime.h"
#include "kernels.hpp"

#include <cassert>
#include <cstdio>
#include <nppcore.h>
#include <nppdefs.h>
#include <nppi_statistics_functions.h>
#include <type_traits>

#include "defs.h"

#define GAMMA 2

/*  __     __   .______           ___        ___   .______   
 * /_ |   / /   |   _  \          \  \      / _ \  |   _  \  
 *  | |  / /_   |  |_)  |     _____\  \    | (_) | |  |_)  | 
 *  | | | '_ \  |   _  <     |______>  >    > _ <  |   _  <  
 *  | | | (_) | |  |_)  |          /  /    | (_) | |  |_)  | 
 *  |_|  \___/  |______/          /__/      \___/  |______/  
*/
__global__ void kernel_convert_16_8(uint16_t *in, uint8_t *out, size_t count, float scale) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position >= count) {
    return;
  }

        float normalized = __saturatef(in[position] / scale);
#ifdef GAMMA
        normalized = pow(normalized, GAMMA);
#endif
        out[position] = normalized * 255;
}

static struct {
        struct {
                void *data;
                int len;
                void *d_res;
        } stat[2];
} state;
/// indices to state.scratch
enum {
        STDDEV_MEAN = 0,
        MAX = 1,
};

template <typename T> struct second_param;
template <typename Ret, typename T1, typename T2, typename... Args>
struct second_param<Ret(T1, T2, Args...)> {
        using type = T2;
};
// Helper to deduce the function type
template <typename Func>
using size_param_t = std::remove_pointer_t<typename second_param<Func>::type>;

void convert_16_8_normalize_cuda(struct dec_image *in, uint8_t *out, hipStream_t stream)
{
        if (nppGetStream() != stream) {
                nppSetStream(stream);
        }
        // NppStreamContext NppStreamContext;
        // rc = nppGetStreamContext(&NppStreamContext);
        // assert(rc == 0);
        NppiSize ROI;
        ROI.width = in->width;
        ROI.height = in->height;

        // int in NPP 12.3 while size_t in 12.6
        size_param_t<decltype(nppiMeanStdDevGetBufferHostSize_16u_C1R)>
            stddev_scratch_len_req = 0;
        size_param_t<decltype(nppiMeanStdDevGetBufferHostSize_16u_C1R)>
            max_scratch_len_req = 0;

        // GetBufferHostSize_16s_C1R_Ctx(ROI, &BufferSize, NppStreamContext);
        nppiMeanStdDevGetBufferHostSize_16u_C1R(ROI, &stddev_scratch_len_req);
        if ((int)stddev_scratch_len_req > state.stat[STDDEV_MEAN].len) {
                hipHostMalloc((void **)(&state.stat[STDDEV_MEAN].data),
                               stddev_scratch_len_req);
                state.stat[STDDEV_MEAN].len = (int)stddev_scratch_len_req;
        }
        nppiMaxGetBufferHostSize_16u_C1R(ROI, &max_scratch_len_req);
        if ((int)max_scratch_len_req > state.stat[MAX].len) {
                hipHostMalloc((void **)(&state.stat[MAX].data),
                               max_scratch_len_req);
                state.stat[MAX].len = (int)max_scratch_len_req;
        }
        // printf("%d\n", BufferSize);
        if (state.stat[STDDEV_MEAN].d_res == nullptr) {
                hipMalloc((void **)(&state.stat[STDDEV_MEAN].d_res),
                           2 * sizeof(Npp64f));
        }
        if (state.stat[MAX].d_res == nullptr) {
                hipMalloc((void **)(&state.stat[MAX].d_res), sizeof(Npp16u));
        }
        NppStatus rc = NPP_NO_ERROR;
        rc = nppiMean_StdDev_16u_C1R(
            (Npp16u *)in->data, ROI.width * 2, ROI,
            (Npp8u *)state.stat[STDDEV_MEAN].data,
            &((Npp64f *)state.stat[STDDEV_MEAN].d_res)[0],
            &((Npp64f *)state.stat[STDDEV_MEAN].d_res)[1]);
        assert(rc == 0);
        Npp64f stddev_mean_res[2];
        hipMemcpyAsync(stddev_mean_res, state.stat[STDDEV_MEAN].d_res, sizeof stddev_mean_res, hipMemcpyDeviceToHost, stream);

        rc = nppiMax_16u_C1R((Npp16u *)in->data, ROI.width * 2, ROI,
                                     (Npp8u*)state.stat[MAX].data, (Npp16u*)state.stat[MAX].d_res);
        assert(rc == 0);
        Npp16u max_res = 0;
        hipMemcpyAsync(&max_res, state.stat[MAX].d_res, sizeof max_res, hipMemcpyDeviceToHost, stream);

        if (log_level >= 1) {
                printf("MEAN: %f STDDEV: %f MAX: %hu\n", stddev_mean_res[0],
                       stddev_mean_res[1], max_res);
        }

        const size_t count = (size_t)in->width * in->height;
        // scale to 0..\mu+2*\sigma
        float scale = MIN(stddev_mean_res[0] + 2 * stddev_mean_res[1], max_res);
        kernel_convert_16_8<<<dim3((count + 255) / 256), dim3(256), 0,
                              stream>>>((uint16_t *)in->data, out, count,
                                        scale);
}

/*                             _                 __    _  __   _     
 *    ___ ___  _ __ ___  _ __ | | _____  __      \ \  / |/ /_ | |__  
 *   / __/ _ \| '_ ` _ \| '_ \| |/ _ \ \/ /  _____\ \ | | '_ \| '_ \ 
 *  | (_| (_) | | | | | | |_) | |  __/>  <  |_____/ / | | (_) | |_) |
 *   \___\___/|_| |_| |_| .__/|_|\___/_/\_\      /_/  |_|\___/|_.__/ 
 *                      |_|                                          
*/
__global__ void kernel_convert_complex_int(const int16_t *in, uint16_t *out,
                                           size_t datalen)
{
        unsigned int position =
            threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
        if (position >= datalen) {
                return;
        }
        out[position] = sqrt(pow(in[2 * position], 2) + pow(in[2 * position + 1], 2));
}
void convert_complex_int_to_uint16(const int16_t *in, uint16_t *out,
                                  size_t count, hipStream_t stream)
{
        kernel_convert_complex_int<<<dim3((count + 255) / 256), dim3(256), 0,
                                     stream>>>(in, out, count);
}


__global__ void kernel_convert_rgba_grayscale(uint8_t *in, uint8_t *out, size_t datalen) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position > datalen) {
    return;
  }
  out[position] = in[position * 4];
}

void convert_rgba_grayscale(uint8_t *in, uint8_t *out, size_t pix_count,
                            void *stream)
{
        kernel_convert_rgba_grayscale<<<dim3((pix_count + 255) / 256),
                                        dim3(256), 0, (hipStream_t)stream>>>(
            in, out, pix_count);
}

__global__ void kernel_convert_rgba_rgb(uint8_t *in, uint8_t *out, size_t datalen) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position >= datalen) {
    return;
  }
  out[position * 3] = in[position * 4];
  out[position * 3 + 1] = in[position * 4 + 1];
  out[position * 3 + 1] = in[position * 4 + 1];
}

void convert_rgba_rgb(uint8_t *in, uint8_t *out, size_t pix_count,
                            void *stream)
{
        kernel_convert_rgba_rgb<<<dim3((pix_count + 255) / 256), dim3(256), 0,
                                  (hipStream_t)stream>>>(in, out, pix_count);
}

template<typename t>
__global__ void kernel_convert_remove_pitch(t *in, t *out,
                                            int width, int spitch)
{
        int position_x = threadIdx.x + blockIdx.x * blockDim.x;
        if (position_x >= width) {
                return;
        }
        int position_y = threadIdx.y + blockIdx.y * blockDim.y;
        out[position_y * width + position_x] =
            in[position_y * spitch + position_x];
}

/**
 * This function is not 100% necessary since GPUJPEG supports pitched
 * input (but currently just RGB) but it won't perhaps dealinkg with it since
 * CUDA kernels are quite fast
 */
void convert_remove_pitch(uint8_t *in, uint8_t *out, int width, int spitch,
                          int height, void *stream)
{
        kernel_convert_remove_pitch<uint8_t><<<dim3((width + 255) / 256, height),
                                      dim3(256), 0, (hipStream_t)stream>>>(
            in, out, width, spitch);
}


/**
 * This function is not 100% necessary since GPUJPEG supports pitched
 * input (but currently just RGB) but it won't perhaps dealinkg with it since
 * CUDA kernels are quite fast
 */
void convert_remove_pitch_16(uint16_t *in, uint16_t *out, int width, int spitch,
                          int height, void *stream)
{
        kernel_convert_remove_pitch<uint16_t><<<dim3((width + 255) / 256, height),
                                      dim3(256), 0, (hipStream_t)stream>>>(
            in, out, width, spitch / 2);
}

void cleanup_cuda_kernels()
{
        for (unsigned i = 0; i < ARR_SIZE(state.stat); ++i) {
                hipHostFree(state.stat[i].data);
                hipFree(state.stat[i].d_res);
        }
}
