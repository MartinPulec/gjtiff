#include "hip/hip_runtime.h"
#include "kernels.hpp"

#include <cassert>
#include <nppcore.h>
#include <nppdefs.h>
#include <nppi_statistics_functions.h>
#include <printf.h>

#include "defs.h"

#define GAMMA 2

/*  __     __   .______           ___        ___   .______   
 * /_ |   / /   |   _  \          \  \      / _ \  |   _  \  
 *  | |  / /_   |  |_)  |     _____\  \    | (_) | |  |_)  | 
 *  | | | '_ \  |   _  <     |______>  >    > _ <  |   _  <  
 *  | | | (_) | |  |_)  |          /  /    | (_) | |  |_)  | 
 *  |_|  \___/  |______/          /__/      \___/  |______/  
*/
__global__ void kernel_convert_16_8(uint16_t *in, uint8_t *out, size_t count, float scale) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position > count) {
    return;
  }

        float normalized = __saturatef(in[position] / scale);
#ifdef GAMMA
        normalized = pow(normalized, GAMMA);
#endif
        out[position] = normalized * 255;
}

static struct {
        Npp64f *d_res;
        Npp8u *scratch;
        int scratch_len;

} state;

void convert_16_8_cuda(struct dec_image *in, uint8_t *out, hipStream_t stream)
{
        if (nppGetStream() != stream) {
                nppSetStream(stream);
        }
        // NppStreamContext NppStreamContext;
        // rc = nppGetStreamContext(&NppStreamContext);
        // assert(rc == 0);
        NppiSize ROI;
        ROI.width = in->width;
        ROI.height = in->height;
        int scratch_len_req = 0;
        // GetBufferHostSize_16s_C1R_Ctx(ROI, &BufferSize, NppStreamContext);
        nppiMeanStdDevGetBufferHostSize_16u_C1R(ROI, &scratch_len_req);
        if (scratch_len_req > state.scratch_len) {
                hipHostMalloc((void **)(&state.scratch), scratch_len_req);
                state.scratch_len = scratch_len_req;
        }
        // printf("%d\n", BufferSize);
        if (state.d_res == nullptr) {
                hipMalloc((void **)(&state.d_res), 2 * sizeof(Npp64f));
        }
        NppStatus rc = NPP_NO_ERROR;
        rc = nppiMean_StdDev_16u_C1R((Npp16u *)in->data, ROI.width * 2, ROI,
                                     state.scratch, &state.d_res[0], &state.d_res[1]);
        assert(rc == 0);
        Npp64f res[2];
        hipMemcpyAsync(res, state.d_res, sizeof res, hipMemcpyDeviceToHost, stream);
        if (log_level >= 1) {
                printf("MEAN: %f STDDEV: %f\n", res[0], res[1]);
        }

        const size_t count = (size_t)in->width * in->height;
        // scale to 0..\mu+2*\sigma
        kernel_convert_16_8<<<dim3((count + 255) / 256), dim3(256), 0,
                              stream>>>((uint16_t *)in->data, out, count,
                                        res[0] + 2 * res[1]);
}

__global__ void kernel_convert_complex_int(const uint8_t *in, uint8_t *out,
                                           size_t datalen)
{
        unsigned int position =
            threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
        if (position > datalen) {
                return;
        }
        out[position] = in[4 * position + 1]; // take just MSB from real part
}
void convert_complex_int(const uint8_t *in, uint8_t *out, size_t in_len,
                         hipStream_t stream)
{
        const size_t count = in_len / 4;
        kernel_convert_complex_int<<<dim3((count + 255) / 256), dim3(256), 0,
                                     stream>>>(in, out, count);
}


__global__ void kernel_convert_rgba_grayscale(uint8_t *in, uint8_t *out, size_t datalen) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position > datalen) {
    return;
  }
  out[position] = in[position * 4];
}

void convert_rgba_grayscale(uint8_t *in, uint8_t *out, size_t pix_count,
                            void *stream)
{
        kernel_convert_rgba_grayscale<<<dim3((pix_count + 255) / 256),
                                        dim3(256), 0, (hipStream_t)stream>>>(
            in, out, pix_count);
}

__global__ void kernel_convert_rgba_rgb(uint8_t *in, uint8_t *out, size_t datalen) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position > datalen) {
    return;
  }
  out[position * 3] = in[position * 4];
  out[position * 3 + 1] = in[position * 4 + 1];
  out[position * 3 + 1] = in[position * 4 + 1];
}

void convert_rgba_rgb(uint8_t *in, uint8_t *out, size_t pix_count,
                            void *stream)
{
        kernel_convert_rgba_rgb<<<dim3((pix_count + 255) / 256), dim3(256), 0,
                                  (hipStream_t)stream>>>(in, out, pix_count);
}

void cleanup_cuda_kernels()
{
        hipHostFree(state.scratch);
        hipFree(state.d_res);
}
