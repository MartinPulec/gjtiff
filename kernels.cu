#include "hip/hip_runtime.h"
#include "kernels.hpp"

__global__ void kernel_convert_16_8(uint16_t *in, uint8_t *out, size_t datalen) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position > datalen) {
    return;
  }
  out[position] = in[position] / 256;
}

void convert_16_8_cuda(uint16_t *in,uint8_t *out, size_t in_len, hipStream_t stream) {
  const size_t count = in_len / 2;
  kernel_convert_16_8<<<dim3((in_len+255)/256), dim3(256), 0, stream>>>(in, out, count);
}


__global__ void kernel_convert_complex_int(const uint8_t *in, uint8_t *out,
                                           size_t datalen)
{
        unsigned int position =
            threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
        if (position > datalen) {
                return;
        }
        out[position] = in[4 * position + 1]; // take just MSB from real part
}
void convert_complex_int(const uint8_t *in, uint8_t *out, size_t in_len,
                         hipStream_t stream)
{
        const size_t count = in_len / 4;
        kernel_convert_complex_int<<<dim3((count + 255) / 256), dim3(256), 0,
                                     stream>>>(in, out, count);
}


__global__ void kernel_convert_rgba_grayscale(uint8_t *in, uint8_t *out, size_t datalen) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position > datalen) {
    return;
  }
  out[position] = in[position * 4];
}

void convert_rgba_grayscale(uint8_t *in, uint8_t *out, size_t pix_count,
                            void *stream)
{
        kernel_convert_rgba_grayscale<<<dim3((pix_count + 255) / 256),
                                        dim3(256), 0, (hipStream_t)stream>>>(
            in, out, pix_count);
}

__global__ void kernel_convert_rgba_rgb(uint8_t *in, uint8_t *out, size_t datalen) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position > datalen) {
    return;
  }
  out[position * 3] = in[position * 4];
  out[position * 3 + 1] = in[position * 4 + 1];
  out[position * 3 + 1] = in[position * 4 + 1];
}

void convert_rgba_rgb(uint8_t *in, uint8_t *out, size_t pix_count,
                            void *stream)
{
        kernel_convert_rgba_rgb<<<dim3((pix_count + 255) / 256), dim3(256), 0,
                                  (hipStream_t)stream>>>(in, out, pix_count);
}
