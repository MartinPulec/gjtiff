#include "hip/hip_runtime.h"
#include "kernels.hpp"

__global__ void kernel_convert_16_8(uint16_t *in, uint8_t *out, size_t datalen) {
  int position = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
  if (position > datalen) {
    return;
  }
  out[position] = in[position] / 256;
}

void convert_16_8_cuda(uint16_t *in,uint8_t *out, size_t in_len, hipStream_t stream) {
  const size_t count = in_len / 2;
  kernel_convert_16_8<<<dim3((in_len+255)/256), dim3(256), 0, stream>>>(in, out, count);
}
